#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cuda_example_1.cu
 Author      : me
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>


static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

__host__ __device__ void func() {
#if __CUDA_ARCH__ >= 600
   // Device code path for compute capability 6.x
	printf("Device code path for compute capability 6.x");
#elif __CUDA_ARCH__ >= 500
   // Device code path for compute capability 5.x
	printf("Device code path for compute capability 5.x");
#elif __CUDA_ARCH__ >= 300
   // Device code path for compute capability 3.x
	printf("Device code path for compute capability 3.x");
#elif __CUDA_ARCH__ >= 200
   // Device code path for compute capability 2.x
	printf("Device code path for compute capability 2.x");
#elif !defined(__CUDA_ARCH__)
   // Host code path
	//printf("Host code path");
#endif
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

float *cpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
	return rc;
}


void initialize(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5*(i+1);
}

//https://gist.github.com/qfgaohao/0a285941c38cceb186fcaa464b349320
/*
 * Device number: 0
  Device name: GeForce GTX 1050
  Compute capability: 6.1

  Clock Rate: 1493000 kHz
  Total SMs: 5
  Shared Memory Per SM: 98304 bytes
  Registers Per SM: 65536 32-bit
  Max threads per SM: 2048
  L2 Cache Size: 524288 bytes
  Total Global Memory: 4238737408 bytes
  Memory Clock Rate: 3504000 kHz

  Max threads per block: 1024
  Max threads in X-dimension of block: 1024
  Max threads in Y-dimension of block: 1024
  Max threads in Z-dimension of block: 64

  Max blocks in X-dimension of grid: 2147483647
  Max blocks in Y-dimension of grid: 65535
  Max blocks in Z-dimension of grid: 65535

  Shared Memory Per Block: 49152 bytes
  Registers Per Block: 65536 32-bit
  Warp size: 32
 */
void deviceQuery ()
{
  hipDeviceProp_t prop;
  int nDevices=0, i;
  hipError_t ierr;

  ierr = hipGetDeviceCount(&nDevices);
  if (ierr != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierr)); }



  for( i = 0; i < nDevices; ++i )
  {
     ierr = hipGetDeviceProperties(&prop, i);
     printf("Device number: %d\n", i);
     printf("  Device name: %s\n", prop.name);
     printf("  Compute capability: %d.%d\n\n", prop.major, prop.minor);

     printf("  Clock Rate: %d kHz\n", prop.clockRate);
     printf("  Total SMs: %d \n", prop.multiProcessorCount);
     printf("  Shared Memory Per SM: %lu bytes\n", prop.sharedMemPerMultiprocessor);
     printf("  Registers Per SM: %d 32-bit\n", prop.regsPerMultiprocessor);
     printf("  Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
     printf("  L2 Cache Size: %d bytes\n", prop.l2CacheSize);
     printf("  Total Global Memory: %lu bytes\n", prop.totalGlobalMem);
     printf("  Memory Clock Rate: %d kHz\n\n", prop.memoryClockRate);


     printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
     printf("  Max threads in X-dimension of block: %d\n", prop.maxThreadsDim[0]);
     printf("  Max threads in Y-dimension of block: %d\n", prop.maxThreadsDim[1]);
     printf("  Max threads in Z-dimension of block: %d\n\n", prop.maxThreadsDim[2]);

     printf("  Max blocks in X-dimension of grid: %d\n", prop.maxGridSize[0]);
     printf("  Max blocks in Y-dimension of grid: %d\n", prop.maxGridSize[1]);
     printf("  Max blocks in Z-dimension of grid: %d\n\n", prop.maxGridSize[2]);

     printf("  Shared Memory Per Block: %lu bytes\n", prop.sharedMemPerBlock);
     printf("  Registers Per Block: %d 32-bit\n", prop.regsPerBlock);
     printf("  Warp size: %d\n\n", prop.warpSize);

  }
}


int main(void)
{
    deviceQuery();

	static const int WORK_SIZE = 65530;
	float *data = new float[WORK_SIZE];

	initialize (data, WORK_SIZE);

	float *recCpu = cpuReciprocal(data, WORK_SIZE);
	float *recGpu = gpuReciprocal(data, WORK_SIZE);
	float cpuSum = std::accumulate (recCpu, recCpu+WORK_SIZE, 0.0);
	float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);

	/* Verify the results */
	std::cout<<"gpuSum = "<<gpuSum<< " cpuSum = " <<cpuSum<<std::endl;

	/* Free memory */
	delete[] data;
	delete[] recCpu;
	delete[] recGpu;

	func();

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

